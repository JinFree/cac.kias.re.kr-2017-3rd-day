
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 4096 * 1024

void saxpy(int n, float a, float *x, float *y){
    for( int i=0; i<n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
    return ;
}

void saxpy_gpu(int n, float a, float *x, float *y){

    //TODO
    //
    return ;
}

int main(){

    float *x, *y;
    float a;
    int size = N * sizeof( float);
    x = (float *) malloc( size);
    y = (float *) malloc( size);

    a=3;

   // initialize for
    for( int i=0; i<N; i++){
      x[i]=i*0.01;
      y[i]=0;
    }

    printf(" data\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    saxpy(N, a, x, y);   // cpu
    saxpy_gpu(N,a,x,y);   // gpu

    printf(" result\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    free(x);
    free(y);

    return 0;
}

