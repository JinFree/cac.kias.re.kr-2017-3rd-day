
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 4096 * 1024

void saxpy(int n, float a, float *x, float *y){
    for( int i=0; i<n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
    return ;
}
__global__ void _dev_saxpy()
{
    return;
}
void saxpy_line6_gpu(int n, float a, float *x, float *y){
    float *x_dev;
    float *y_dev;
    int size = N * sizeof(float);
    hipMalloc((void**)&x_dev, size);
    hipMalloc((void**)&y_dev, size);
    hipMemcpy(x_dev, x, size, hipMemcpyHostToDevice);
    hipMemcpy(y_dev, y, size, hipMemcpyHostToDevice);   

return ;
} 

int main(){

    float *x, *y;
    float a;
    int size = N * sizeof( float);
    x = (float *) malloc( size);
    y = (float *) malloc( size);

    a=3;

   // initialize for
    for( int i=0; i<N; i++){
      x[i]=2;
      y[i]=0;
    }

    printf(" data\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    saxpy_line6_gpu(N, a, x, y);

    printf(" result\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    free(x);
    free(y);

    return 0;
}

