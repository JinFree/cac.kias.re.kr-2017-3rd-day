
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 4096 * 1024

void saxpy(int n, float a, float *x, float *y){
    for( int i=0; i<n; i++)
    {
        y[i] = a * x[i] + y[i];
    }
    return ;
}

void saxpy_line6_gpu(int n, float a, float *x, float *y){
  float *x_gpu;
  float *y_gpu;
  int size = N * sizeof( float);
  hipMalloc( (void**)& x_gpu, size );
  hipMalloc( (void**)& y_gpu, size );

  hipMemset( x_gpu, 0.0 , size);
  hipMemset( y_gpu, 0.0 , size);

  hipMemcpy( x_gpu, x, size , hipMemcpyHostToDevice);
  hipMemcpy( y_gpu, y, size , hipMemcpyHostToDevice);
 
//TODO kernel 

  hipMemcpy( y, y_gpu, size , hipMemcpyDeviceToHost);

  hipFree(x_gpu);
  hipFree(y_gpu); 

return ;
} 

int main(){

    float *x, *y;
    float a;
    int size = N * sizeof( float);
    x = (float *) malloc( size);
    y = (float *) malloc( size);

    a=3;

   // initialize for
    for( int i=0; i<N; i++){
      x[i]=2;
      y[i]=0;
    }

    printf(" data\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    saxpy_line6_gpu(N, a, x, y);

    printf(" result\n");
    for( int i = 0; i < 5; ++i )  printf("y[%d] = %f, ", i, y[i]);
    printf ("\n");

    free(x);
    free(y);

    return 0;
}

